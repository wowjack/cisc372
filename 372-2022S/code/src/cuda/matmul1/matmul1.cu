/* matmul1.cu:  simple CUDA version of matrix-matrix multiplication.
   Command line args are N, L, M.  A is NxL, B is LxM, C is NxM. */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <sys/time.h>

/* Number of threads per block in x and y direction. 32*32=1024 */
const int nthreadsx = 32, nthreadsy = 32;

double mytime() {
  struct timeval t;
  gettimeofday(&t, NULL);
  return t.tv_sec + t.tv_usec/1000000.0;
}

/* Allocate n*m doubles in the host's heap */
double * cpuAlloc(int n, int m) {
  double * result = (double*)malloc(n*m*sizeof(double));
  assert(result);
  return result;
}

/* Allocate n*m doubles on the device global memory */
double * gpuAlloc(int n, int m) {
  double * result;
  int err = hipMalloc(&result, n*m*sizeof(double));
  assert(err == hipSuccess);
  return result;
}

/* Print matrix mat, which has numRows rows and numCols cols */
void printMatrix(int numRows, int numCols, double * mat) {
  for (int i = 0; i < numRows; i++) {
    for (int j = 0; j < numCols; j++)
      printf("%6.1f ", mat[i*numCols + j]);
    printf("\n");
  }
  printf("\n");
}

/* Kernel.  Multiplies a and b, sticking results into c.
   a is nxl, b is lxm, c is nxm. */
__global__ void multiply(int n, int l, int m,
			 double * a, double * b, double * c) {
  int i = blockDim.y * blockIdx.y + threadIdx.y;
  int j = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < n && j < m) {
    double result = 0.0;
    for (int k = 0; k < l; k++)
      result += a[i*l + k] * b[k*m + j]; // a[i][k] * b[k][j];
    c[i*m + j] = result; // c[i][j]
  }
}

int main(int argc, char * argv[]) {
  assert(argc == 4);

  int err, N = atoi(argv[1]), L = atoi(argv[2]), M = atoi(argv[3]);

  assert(N>=1); assert(L>=1); assert(M>=1);

  dim3 blockDim(nthreadsx, nthreadsy),
    gridDim(M/nthreadsx + (M % nthreadsx != 0),
	    N/nthreadsy + (N % nthreadsy != 0));
  double * a = cpuAlloc(N, L), * b = cpuAlloc(L, M), * c = cpuAlloc(N, M),
    * a_d = gpuAlloc(N, L), * b_d = gpuAlloc(L, M), * c_d = gpuAlloc(N, M);

  printf("matmul1.cu: N=%d, L=%d, M=%d, gridDim=(%d,%d), blockDim=(%d,%d)\n",
	 N, L, M, gridDim.x, gridDim.y, blockDim.x, blockDim.y);
  for (int i = 0; i < N; i++)
    for (int j = 0; j < L; j++)
      a[i*L + j] = rand()*1.0/RAND_MAX;
  for (int i = 0; i < L; i++)
    for (int j = 0; j < M; j++)
      b[i*M + j] = rand()*1.0/RAND_MAX;
#ifdef DEBUG
  printMatrix(N, L, a);
  printMatrix(L, M, b);
#endif
  printf("Starting computation.\n"); fflush(stdout);
  double time = mytime();
  err = hipMemcpy(a_d, a, N*L*sizeof(double), hipMemcpyHostToDevice);
  assert(err == hipSuccess);
  err = hipMemcpy(b_d, b, L*M*sizeof(double), hipMemcpyHostToDevice);
  assert(err == hipSuccess);
  multiply<<<gridDim,blockDim>>>(N, L, M, a_d, b_d, c_d);
  err = hipMemcpy(c, c_d, N*M*sizeof(double), hipMemcpyDeviceToHost);
  assert(err == hipSuccess);
#ifdef DEBUG
  printMatrix(N, M, c);
#endif
  printf("Done.  Time = %lf.\n", mytime() - time);
}
